#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <optix.h>
#include <optix_math.h>
#include <math.h>
#include "Light.hpp"
#include "Photon.hpp"
#include "random.hpp"
#include "random_float3.hpp"


struct ShadowPayload {
  float3 attenuation;
  uint   lightIndex;
};

struct WindowPayload {
  uint2 seed;
  uint  windowIndex;
};

struct AreaLightPayload {
  float3 energy;
  uint2  seed;
};

struct SkyPayload {
  float3 energy;
  float3 attenuation;
  float2 angles;
  uint   sky;
};

struct PhotonPayload {
  float3 energy;
  uint2  seed;
  uint   pmIndex;
  uint   numDeposits;
  uint   rayDepth;
  uint   windowIndex;
};
rtBuffer<char,    1>     screenBuffer;
rtBuffer<uint2>                 randBuffer;
rtBuffer<DirectionalLight>      dirLightBuffer;
rtBuffer<PhotonRecord>          skyPhotonBuffer;
rtBuffer<PhotonRecord>          areaLightPhotonBuffer;
rtBuffer<ClearWindowLaunchInfo> windowBuffer;
rtBuffer<AreaLightLaunchInfo>   areaLightBuffer;
rtBuffer<float3,    1>     normalBuffer;

rtDeclareVariable(rtObject,         topObject,       ,                           );
rtDeclareVariable(rtObject,         topShadower,     ,                           );

rtDeclareVariable(PhotonPayload,    photonPayload,   rtPayload,                  );
rtDeclareVariable(WindowPayload,    windowPayload,   rtPayload,                  );
rtDeclareVariable(AreaLightPayload, lightPayload,    rtPayload,                  );
rtDeclareVariable(SkyPayload,       skyPayload,      rtPayload,                  );
rtDeclareVariable(uint,             launchIndex,     rtLaunchIndex,              );
rtDeclareVariable(float3,           shadingNormal,   attribute shading_normal,   ); 
rtDeclareVariable(float3,           geometricNormal, attribute geometric_normal, );
rtDeclareVariable(float3,           centroid,        attribute centroid,         );

rtDeclareVariable(uint,             maxPhotonCount,  ,                           );
rtDeclareVariable(uint,             maxDepth,        ,                           );
rtDeclareVariable(float2,           sunAngles,       ,                           );

rtDeclareVariable(uint,             shadowRayType,   ,                           );
rtDeclareVariable(uint,             windowRayType,   ,                           );
rtDeclareVariable(uint,             lightRayType,    ,                           );
rtDeclareVariable(uint,             skyRayType,      ,                           );
rtDeclareVariable(uint,             photonRayType,   ,                           );
rtDeclareVariable(float,            sceneEpsilon,    ,                           );
rtDeclareVariable(float,            sunBrightness,    ,                           );
rtDeclareVariable(float3,            sunDirection,    ,                           );
rtDeclareVariable(int,              normalInversion, ,                           );
rtDeclareVariable(int,          bounce, ,                           );
// Material properties
rtDeclareVariable(float3,        Kd,              ,                           );
rtDeclareVariable(float3,        Ks,              ,                           );
rtDeclareVariable(float3,        Td,              ,                           );
rtDeclareVariable(float3,        Ts,              ,                           );

rtDeclareVariable(optix::Ray,    currentRay,      rtCurrentRay,               );
rtDeclareVariable(float,         hitT,            rtIntersectionDistance,     );
rtDeclareVariable(int,        screen,              ,                           );
rtDeclareVariable(int,        screenWidth,              ,                           );
rtDeclareVariable(int,        screenHeight,              ,                           );
rtDeclareVariable(int,        skyType,              ,                           );

rtDeclareVariable(int,        centroidNumber,    attribute centroidNumber,   );

RT_PROGRAM void photonPassGenerator() {
  uint2 seed = randBuffer[launchIndex];
}

RT_PROGRAM void areaLightPassGenerator() {
  //Initially zeroes out all photons
  for(unsigned int i = 0; i < maxPhotonCount; ++i) {
    areaLightPhotonBuffer[launchIndex + i].energy = make_float3(0);
  }

  
  const AreaLightLaunchInfo& info = areaLightBuffer[launchIndex];
  optix::Ray ray(info.start, info.normal, lightRayType, info.eps, 3 * info.eps);

  // WindowPayload payload;
  // payload.seed        = randBuffer[launchIndex];
  // payload.windowIndex = info.windowIndex;
  // payload.launchIndex = launchIndex;
  AreaLightPayload payload;
  payload.seed   = randBuffer[launchIndex];
  payload.energy = info.energy;
  rtTrace(topObject, ray, payload);
}
RT_PROGRAM void areaLightPassClosestHit() {
  const float3 direction = currentRay.direction;
  const float3 hitPoint  = currentRay.origin + hitT * direction;

  // Perp values will be used later, so might as well calc them
  float3 perpX = cross(direction, make_float3(1, 0, 0));
  if(length(perpX) < 1e-6) {
    perpX = cross(direction, make_float3(0, 1, 0));
  }
  perpX=normalize(perpX);
  const float3 perpZ = normalize(cross(direction, perpX));

  float3 dir;

  float  directOrSky = rnd(lightPayload.seed.x);

  const float u1 = rnd(lightPayload.seed.x);
  const float u2 = rnd(lightPayload.seed.y);
  const float r  = sqrt(u1);
  const float theta = 2 * M_PI * u2;

  const float x = r * cos(theta);
  const float z = r * sin(theta);
  //What distribution is this?
  dir = x * perpX + z * perpZ + sqrt(1 - u1) * normalInversion * direction;

  PhotonPayload photonPayload;
  photonPayload.seed        = lightPayload.seed;
  photonPayload.pmIndex     = launchIndex;
  photonPayload.numDeposits = 0;
  photonPayload.rayDepth    = 1;
  photonPayload.energy      = lightPayload.energy;

  optix::Ray photon(hitPoint, -dir, photonRayType, sceneEpsilon);
  rtTrace(topObject, photon, photonPayload);
}

RT_PROGRAM void windowPassGenerator() {
  for(unsigned int i = 0; i < maxPhotonCount; ++i) {
    skyPhotonBuffer[launchIndex + i].energy = make_float3(0);
  }

   const ClearWindowLaunchInfo& info = windowBuffer[launchIndex];

   optix::Ray ray(info.start, info.normal, windowRayType, info.eps, 3 * info.eps);
   WindowPayload payload;
   payload.seed        = randBuffer[launchIndex];
   payload.windowIndex = info.windowIndex;
   // payload.launchIndex = launchIndex; 
   rtTrace(topObject, ray, payload);

}

// This function fires rays from a random point on the window
// This point has been determined already inside of the CPU
// 
//                |
//        •       |  <---  •
//                |
//       end    window   start
// 
// This function is called when it hits the window, which then
// picks a random direction (hopefully) towards the sky to fire.
// It then reverses the direction of the ray, and fires it back
// inside.
RT_PROGRAM void windowPassClosestHit() {
  const float3 direction = currentRay.direction;
  const float3 hitPoint  = currentRay.origin + hitT * direction;
  int thisRayCounts=1;
  
  
        float3 normal=normalBuffer[centroidNumber];
      float cosX=abs(normal.x);
      float cosY=abs(normal.y);
      float cosZ=abs(normal.z);
        float3 xVec,yVec;
      float maxVal=0;
      if (cosX>cosY)
      {
       
        maxVal=cosX;
        xVec=make_float3(0,1,0);
        yVec=make_float3(0,0,1);

      }
      else
      {
       
        maxVal=cosY;
        xVec=make_float3(1,0,0);
        yVec=make_float3(0,0,1);
      }
      if(cosZ>maxVal)
      {
       
        maxVal=cosZ;
        xVec=make_float3(1,0,0);
        yVec=make_float3(0,1,0);

      }
      int xCoord=(int)20*dot(hitPoint,xVec)*screenWidth;
      int yCoord=(int)20*dot(hitPoint,yVec)*screenHeight;
      xCoord%=screenWidth;
      yCoord%=screenHeight;
  
  if (screen&&screenBuffer[yCoord*screenWidth+xCoord]=='X')
       thisRayCounts=0;
  
  // Perp values will be used later, so might as well calc them
  float3 perpX = normalize(cross(direction, make_float3(1, 0, 0)));
  if(length(perpX) < 1e-6) {
    perpX = normalize(cross(direction, make_float3(0, 1, 0)));
  }
  const float3 perpZ = normalize(cross(direction, perpX));
  float3 dir;

  float  directOrSky = rnd(windowPayload.seed.x);
  SkyPayload payload;


  if(directOrSky < 0.5) {
    // SKY
    const float u1 = rnd(windowPayload.seed.x);
    const float u2 = rnd(windowPayload.seed.y);
    const float theta = 2 * M_PI * u2;
    const float r  = sqrt(u1);

    const float x = r * cos(theta);
    const float z = r * sin(theta);
    dir = x * perpX + z * perpZ + sqrt(1 - u1) * normalInversion * -direction;

    float3 ndir = normalize(dir);
    //Dir Zenith is from the straight out angle.  (Opposite of CIE paper)
    float  dirZenith  = acos(ndir.y);
    float  dirAzimuth = atan2(ndir.x, ndir.z);

    payload.sky = 1;
    payload.attenuation = make_float3(1);
    payload.angles = make_float2(dirZenith, dirAzimuth);
    //For indirect sun, no sky
    if(bounce==2||bounce==6)
      thisRayCounts=0;
    //  payload.attenuation=make_float3(0);
  } else {
    // DIRECT SUN
    const DirectionalLight& light = dirLightBuffer[0];
    dir = -light.direction;
    payload.sky = 0;
    payload.attenuation = make_float3(dot(dir, direction));
    //For direct sky and indirect sky, no sun.
    if(bounce>3&&bounce!=6)
        thisRayCounts=0;
//        payload.attenuation = make_float3(0);
  }

  payload.energy = make_float3(0);
  // payload.seed   = windowPayload.seed;
  // payload.depth  = 0;
  optix::Ray ray(hitPoint, dir, skyRayType, sceneEpsilon);
  rtTrace(topObject, ray, payload);
  payload.energy *= payload.attenuation*thisRayCounts;

  if(fmaxf(payload.energy) > 0) {
    PhotonPayload photonPayload;
    photonPayload.seed        = windowPayload.seed;
    photonPayload.pmIndex     = launchIndex;
    photonPayload.numDeposits = 0;
    photonPayload.rayDepth    = payload.sky;
    photonPayload.windowIndex = windowPayload.windowIndex;

    const float avgTd = (Td.x + Td.y + Td.z) / 3;
    const float avgTs = (Ts.x + Ts.y + Ts.z) / 3;
    const float sumT  = avgTd + avgTs;
    const float roll  = rnd(photonPayload.seed.y);

    if(roll < sumT) {
      if(roll < avgTs) {
        photonPayload.energy = payload.energy * Ts / avgTs;
        optix::Ray photon(hitPoint, -dir, photonRayType, sceneEpsilon);
        rtTrace(topObject, photon, photonPayload);
      } else {
        photonPayload.energy = payload.energy * Td / avgTd;

        const float u1 = rnd(photonPayload.seed.x);
        const float u2 = rnd(photonPayload.seed.y);
        const float r  = sqrt(u1);
        const float theta = 2 * M_PI * u2;

        const float x = r * cos(theta);
        const float z = r * sin(theta);
        const float3 newRayDir = x * perpX + z * perpZ + sqrt(1 - u1) * normalInversion * direction;



        optix::Ray photon(hitPoint, newRayDir, photonRayType, sceneEpsilon);
        rtTrace(topObject, photon, photonPayload);
      }
    }
  }
}

// Photon hit logic
RT_PROGRAM void photonClosestHit() {
  //rtPrintf("photon hit\n");
  const float3 direction         = currentRay.direction;
  const float3 hitPoint          = currentRay.origin + hitT * direction;
  const float3 wsShadingNormal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
  const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
  const float3 ffnormal          = faceforward(wsShadingNormal, -direction, wsGeometricNormal);

  if(fmaxf(Kd) > 0) {
    // Diffuse, so store a photon
    if((photonPayload.rayDepth > 0||bounce==6) &&
     (bounce!=5||photonPayload.rayDepth!=1))
    //If it was sky the ray depth is already 1
    //If it's sun we disregard because direct light was already calculated
     {
      PhotonRecord& rec = skyPhotonBuffer[photonPayload.pmIndex + photonPayload.numDeposits];
      rec.position    = hitPoint;
      rec.windowIndex = photonPayload.windowIndex;
      rec.normal      = ffnormal;
      rec.rayDir      = currentRay.direction;
      rec.energy      = photonPayload.energy;
      ++photonPayload.numDeposits;
    }
  }

  if(photonPayload.numDeposits >= maxPhotonCount ||
     photonPayload.rayDepth    >= maxDepth||
     bounce==4) {
    return;
  }

  ++photonPayload.rayDepth;

  // Russian roulette the bounce
  float4 pr;
  const float avgKd = (Kd.x + Kd.y + Kd.z) / 3;
  const float avgTd = (Td.x + Td.y + Td.z) / 3;
  const float avgKs = (Ks.x + Ks.y + Ks.z) / 3;
  const float avgTs = (Ts.x + Ts.y + Ts.z) / 3;
  //rtPrintf("Kd %f Td %f Ks %f Ts %f \n", avgKd, avgTd, avgKs, avgTs);
  pr.x = avgKd;
  pr.y = pr.x + avgTd;
  pr.z = pr.y + avgKs;
  pr.w = pr.z + avgTs;

  float3 rayDir;
  const float roulette = rnd(photonPayload.seed.x);
  //Kd or Td
  if(roulette < pr.y) {
    // Diffuse
    
    //Random number
    //const float u1 = rnd(photonPayload.seed.x);
    //const float u2 = rnd(photonPayload.seed.y);
 
    rayDir=diffuse_sample_point(ffnormal,photonPayload.seed.x,photonPayload.seed.y);
    //rayDir =  specular_sample_point(ffnormal, direction, .02, 
    //          photonPayload.seed.x,photonPayload.seed.y);
    //reflect(direction, ffnormal);
    
    //Kd
    if(roulette < pr.x) {
      // Reflection
      photonPayload.energy *= (Kd / avgKd);
    } 
    //Td
    else 
    {
      // Transmission
      rayDir = -rayDir;
      photonPayload.energy *= (Td / avgTd);
    }
  } 
  // Ks or Ts
  else if(roulette < pr.w) 
  {
    
    // Specular ks
    if(roulette < pr.z) {
      // Reflection
      rayDir = reflect(direction, ffnormal);
      photonPayload.energy *= (Ks / avgKs);
    } 
    // Ts
    else 
    {
      // Transmission
      // Because we're assuming no refraction or complex BTDFs, we do this
      // to not deposit photons in certain cases
      --photonPayload.rayDepth;
      rayDir = direction;
      photonPayload.energy *= (Ts / avgTs);
    }
  } else {
    // Absorb
    return;
  }
  //float3 hacked_start_point = make_float3(0,.1,0);
  optix::Ray bounceRay(hitPoint, rayDir, photonRayType, sceneEpsilon);
  rtTrace(topObject, bounceRay, photonPayload);
}

RT_PROGRAM void skyPassMiss() {
  if(skyPayload.sky) {
    // Sky illumination

    if(skyPayload.angles.x < M_PI / 2) {
      float chi  = acos(cos(sunAngles.x) * cos(skyPayload.angles.x) +
                        sin(sunAngles.x) * sin(skyPayload.angles.x) * cos(abs(skyPayload.angles.y - sunAngles.y)));
      // float phi0 = 1 + a * exp(b);
      float phiZ;
      float fChi;
      float a,b,c,d,e;
      if(skyType==0)
      {
        a = -1.0;
        b = -0.32;
        c = 10;
        d = -3.0;
        e = 0.45;        
        phiZ = 1 + a * exp(b / cos(skyPayload.angles.x));
        fChi = .91 + c * (exp(d * chi) /*- exp(d * M_PI / 2)*/) + e * cos(chi) * cos(chi);
        skyPayload.energy = make_float3(fChi * phiZ);
      }
      else if(skyType==1)
      {
        a = -1.0;
        b = -0.32;
        c = 16;
        d = -3.0;
        e = 0.3;
        phiZ = 1 + a * exp(b / cos(skyPayload.angles.x));
        fChi = .856 + c * (exp(d * chi) /*- exp(d * M_PI / 2)*/) + e * cos(chi) * cos(chi);
        skyPayload.energy = make_float3(fChi * phiZ);
      }
      else if(skyType==2)
      {
        float x=M_PI/2.f-sunAngles.x;
        float y=M_PI/2.f-skyPayload.angles.x;
        a=1.35*(sin(3.59*y-.009)+2.31f)*
          (sin(2.6f*x+.316)+y+4.799f)/2.326;
        b=-.563f*((y+1.059)*(x-.008)+.812);
        skyPayload.energy = make_float3(a*exp(chi*b));
      }
      else if(skyType==3)
      {
        skyPayload.energy = make_float3((1.f+ 2.f * sin(M_PI/2.f-skyPayload.angles.x))/3.f);
      }
      else
      {
        phiZ = 1.;
        fChi = 1.;
      }


    }
  } else {
    // SUN
    const DirectionalLight& light = dirLightBuffer[0];
    skyPayload.energy = light.color * light.intensity;
  }
}

RT_PROGRAM void skyPassClosestHit() {
  // TODO: Figure out if this is necessary
  const float3 direction         = currentRay.direction;
  const float3 hitPoint          = currentRay.origin + hitT * direction;
  const float3 wsShadingNormal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
  const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
  const float3 ffnormal          = faceforward(wsShadingNormal, -direction, wsGeometricNormal);

  if(fmaxf(Kd) > 0) {
    // Compute direct lighting
    int numLights = dirLightBuffer.size();
    while(numLights--) {
      const DirectionalLight& light = dirLightBuffer[numLights];
      float nDL = dot(ffnormal, -sunDirection);

      if(nDL > 0) {
        ShadowPayload shadowPRD;
        shadowPRD.attenuation = make_float3(1);
        shadowPRD.lightIndex  = numLights;
        optix::Ray shadowRay(hitPoint, -sunDirection, shadowRayType, sceneEpsilon);
        rtTrace(topShadower, shadowRay, shadowPRD);

        skyPayload.energy += Kd * nDL * /*light.color*/sunBrightness * 1/*light.intensity*/ * shadowPRD.attenuation;
      }
    }


  }
}
