#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <optix.h>
#include <optix_math.h>
#include <math.h>

#include "Hit.hpp"
#include "Photon.hpp"
#define DEBUG
#define ALPHA 0.5
#define MAGIC_PHOTON_FACTOR .017
#define MAGIC_NONPHOTON_FACTOR 1.4
#define OVEREXPOSURE_THRESHOLD .03

rtBuffer<PackedPhotonRecord> photonMapBuffer;
rtBuffer<PackedHitRecord, 2> eyeHitBuffer;
rtBuffer<float4,          2> outputBuffer;
rtBuffer<float3,          2> directBuffer;
rtBuffer<float4,          1> intermediateBuffer;
rtDeclareVariable(uint2,         launchDim,       rtLaunchDim,                );
rtDeclareVariable(uint,          totalPhotonsFired, ,                           );
rtDeclareVariable(uint,          photonsPerPass,    ,                           );
rtDeclareVariable(uint2,         launchIndex,       rtLaunchIndex,              );
rtDeclareVariable(float,         windowCorrection,  ,                           );
rtDeclareVariable(int,          bounce, ,                           );
rtDeclareVariable(int,          exPer, ,                           );
rtDeclareVariable(float,          factor, ,                           );
rtDeclareVariable(int,          res,      ,                           );

RT_PROGRAM void gatherPass() {
  //if(launchIndex.x==0&&launchIndex.y==0)
  //rtPrintf("gather pass\n");
  PackedHitRecord hit = eyeHitBuffer[launchIndex];
  const float3 hitPosition  = make_float3(hit.a);
  float        hitRadiusSq  = hit.a.w;

  const float3 hitNormal    = make_float3(hit.b);
  float        hitPhotonCnt = hit.b.w;
  const float3 hitKd        = make_float3(hit.c);
  const float  hitMinRds    = hit.c.w;
  float3       hitFlux      = make_float3(hit.d);
  const float3 hitDirect    = make_float3(hit.e);
  const uint   hitFlags     = __float_as_int(hit.e.w);
    
  //BUG: Find out what this does!!!!
  // TODO: Confirm this does what we think we want it to do
  // If there isn't a hit?
  if(!(hitFlags & HitRecord::HIT)) {
    if (launchDim.y>1)
    {
      outputBuffer[launchIndex] = make_float4(hitKd, 1);
    }
    //else if(res==3)
    //{
    //  intermediateBuffer[launchIndex.x] = make_float4(0,0,0, 1);
    //} 
    else
    {
      intermediateBuffer[launchIndex.x] = make_float4(hitKd, 1);
    }
    return;
  }

#define MAX_DEPTH 100
  unsigned int stack[MAX_DEPTH];
  unsigned int stackSize = 0;
  unsigned int node      = 0;

#define pushNode(N) stack[stackSize++] = (N)
#define popNode()   stack[--stackSize]

  float3 flux = make_float3(0);
  // float  compensationFactor = 1;

  // uint m = 0;

  pushNode(0);
  // unsigned int numNewPhotons = 0;
  unsigned int iterations    = 0;
  
  // Do while navigating kd-tree
  do {
    const PackedPhotonRecord& photon = photonMapBuffer[node];
  
    //Use the axis of the photon
    unsigned int axis = __float_as_int(photon.d.w);
    
    //If the photon record of specified axis is NILL
    if(!(axis & PhotonRecord::NILL)) {
      float3 photonPosition = make_float3(photon.a);
      float3 diff = hitPosition - photonPosition;
      float  distanceSq = dot(diff, diff);

      //If the photon is in the specified distance
      if(distanceSq <= hitRadiusSq) {
        float3 photonEnergy = make_float3(photon.d);
        float3 photonNormal = make_float3(photon.b);
        float  cosTerm      = dot(photonNormal, hitNormal);
        //rtPrintf("cosTerm %f \n", cosTerm);
        //if(cosTerm > 0.001f) { // Fudge factor for imperfect cornell box geom
        //Corrected from eric's.  We should want almost identical vectors (Dot product of 1).
        if(cosTerm > 0.99f) { // Fudge factor for imperfect cornell box geom
          // ++m;
          // flux += hitKd * photonEnergy * cosTerm;

          // float g  = (hitPhotonCnt * ALPHA + ALPHA) / (hitPhotonCnt * ALPHA + 1);
          // compensationFactor *= g;
          // hitRadiusSq *= g;
          // ++hitPhotonCnt;
          // flux      = (flux + hitKd * photonEnergy * cosTerm) * g;
          // hitFlux      = hitFlux + (hitKd * photonEnergy * cosTerm) * g;

          float a    = ALPHA;
          float g    = (hitPhotonCnt + a) / (hitPhotonCnt + 1);
          //g = hitPhotonCt +.5 / hitPhotonCnt + 1 
          float rHat = hitRadiusSq * g;
          if(g>1)
          {
            rtPrintf("shouldnt be here");
            while(1);
          }
          if(rHat < hitMinRds) {
            a = (hitMinRds / hitRadiusSq) * (hitPhotonCnt + 1) - hitPhotonCnt;
            g = (hitPhotonCnt + a) / (hitPhotonCnt + 1);
            rHat = hitMinRds;
            rtPrintf("shouldnt be here");
            while(1);

          }
          hitPhotonCnt += a;
          hitRadiusSq   = rHat;
          //if(hitRadiusSq>100)
            //rtPrintf("hrs %f \n", hitRadiusSq);

          hitFlux       = (hitFlux + hitKd * photonEnergy * cosTerm) * g;
        }// end if cosTerm
      }//end if distanceSq

      if(!(axis & PhotonRecord::LEAF)) {
        float d;
        if     (axis & PhotonRecord::X) d = diff.x;
        else if(axis & PhotonRecord::Y) d = diff.y;
        else                            d = diff.z;

        int child = d < 0 ? 0 : 1;
        if(d * d < hitRadiusSq) {
          pushNode((node << 1) + 2 - child);
        }
        node = (node << 1) + 1 + child;
      } else {
        node = popNode();
      }
    } else {
      node = popNode();
    }
    ++iterations;
  } while(node);

  hit.a.w = hitRadiusSq;
  hit.b.w = hitPhotonCnt;
  hit.d   = make_float4(hitFlux);

  eyeHitBuffer[launchIndex] = hit;

  float3 indirectFlux = 1.0 / (M_PI * hitRadiusSq) * (hitFlux / totalPhotonsFired / photonsPerPass);

  //By default this one (below)
  float3 total ;
  if(factor>0)
  {
   float indirectFactor=1.f;
   float directFactor=1.f;
   
   if(res==3)//Hybrid rendering
      directFactor=0.f;
   
   if(bounce==1)
      indirectFactor=0.f;
   else if(bounce>1)
      directFactor=0.f;
   //total = .5*hitDirect*directFactor
   total = .5*directBuffer[launchIndex]*directFactor*MAGIC_NONPHOTON_FACTOR  
         + .5*indirectFlux * 100 * windowCorrection*factor*indirectFactor*MAGIC_PHOTON_FACTOR;// *distrFactor; 
 /*  if(bounce==0)
      total = .5*hitDirect + .5*indirectFlux * 100 * windowCorrection*factor;// *distrFactor;
   else if(bounce==1)
      total = .5*hitDirect;
   else
      total = .5*indirectFlux * 100 * windowCorrection*factor;// *distrFactor;*/
  }
  else total=make_float3(0);
  // float3 total =  indirectFlux * 100 * windowCorrection;
  // float3 total = hitDirect + indirectFlux * 15 * windowCorrection;
  // float3 total = indirectFlux * 800000;
  // float3 total = hitDirect + indirectFlux * 15000;
  // float a = 0.2125 * total.x + 0.7154 * total.y + 0.0721 * total.z + 1;
  total = .01*make_float3(
    powf(total.x, 1 / 2.4),
    powf(total.y, 1 / 2.4),
    powf(total.z, 1 / 2.4)
  );
  // outputBuffer[launchIndex] = make_float4(total / a);
  if (launchDim.y>1) //If full res (not tri or patches)
  {
   //if(total.x>OVEREXPOSURE_THRESHOLD)
   //  outputBuffer[launchIndex] =  make_float4(1,0,0,1);
   //else
     outputBuffer[launchIndex] = make_float4(exPer*total);
  }
  //else //If not at full res (e.g triangles, patches)
  //{   if(total.x>OVEREXPOSURE_THRESHOLD)
  //   intermediateBuffer[launchIndex.x] =  make_float4(1,0,0,1);
  // else
     intermediateBuffer[launchIndex.x] = make_float4(exPer*total);
  //}
    // intermediateBuffer[launchIndex.x] = make_float4(exPer*total);
  //if(launchIndex.x%100==0)
   // rtPrintf("output %f\n", outputBuffer[launchIndex].x);
}
