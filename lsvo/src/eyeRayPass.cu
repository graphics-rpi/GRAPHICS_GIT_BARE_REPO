#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

#include "Hit.hpp"
#include "Light.hpp"
#include "screen_utils.h"
#include "camera_rays.h"

struct EyePayload {
    int    centroidNumber;
  float3 attenuation;
  float3 direct;
  uint   depth;

};

struct ShadowPayload {
  float3 attenuation;
  uint   lightIndex;

};

rtBuffer<DirectionalLight> dirLightBuffer;
rtBuffer<HitRecord, 2>     eyeHitBuffer;
rtBuffer<float3, 2>    directBuffer;

rtBuffer<float3,    1>     centroidBuffer;
rtBuffer<float3,    1>     normalBuffer;


rtBuffer<char,    1>     screenBuffer;
// Scene graph
rtDeclareVariable(rtObject,      topObject,       ,                           );
rtDeclareVariable(rtObject,      topShadower,     ,                           );

// Camera
rtDeclareVariable(float3,        eye,             ,                           );
rtDeclareVariable(float3,        U,               ,                           );
rtDeclareVariable(float3,        V,               ,                           );
rtDeclareVariable(float3,        W,               ,                           );

rtDeclareVariable(EyePayload,    eyePayload,      rtPayload,                  );
rtDeclareVariable(ShadowPayload, shadowPayload,   rtPayload,                  );
rtDeclareVariable(uint2,         launchIndex,     rtLaunchIndex,              );
rtDeclareVariable(uint2,         launchDim,       rtLaunchDim,                );
rtDeclareVariable(float3,        shadingNormal,   attribute shading_normal,   );
rtDeclareVariable(float3,        geometricNormal, attribute geometric_normal, );

rtDeclareVariable(uint,          eyeRayType,      ,                           );
rtDeclareVariable(uint,          shadowRayType,   ,                           );
rtDeclareVariable(float3,        sceneCenter,     ,                           );
rtDeclareVariable(float,         sceneEpsilon,    ,                           );
rtDeclareVariable(float,         sceneMaxDim,     ,                           );
rtDeclareVariable(uint,          backfaceCulling, ,                           );

// Material properties
rtDeclareVariable(float3,        Kd,              ,                           );
rtDeclareVariable(float3,        Ks,              ,                           );
rtDeclareVariable(float3,        Td,              ,                           );
rtDeclareVariable(float3,        Ts,              ,                           );

rtDeclareVariable(float3,        nearPoint1,              ,                   );
rtDeclareVariable(float3,        nearPoint2,              ,                   );
rtDeclareVariable(float3,        nearPoint3,              ,                   );
rtDeclareVariable(float3,        nearPoint4,              ,                           );
rtDeclareVariable(float3,        cameraDir,               ,                   );
rtDeclareVariable(int,        screenWidth,                ,                   );
rtDeclareVariable(int,        screenHeight,               ,                   );
rtDeclareVariable(int,        screen,                     ,                   );

rtDeclareVariable(optix::Ray,    currentRay,      rtCurrentRay,               );
rtDeclareVariable(float,         hitT,            rtIntersectionDistance,     );
rtDeclareVariable(int,        centroidNumber,    attribute centroidNumber,    );
//debugging
rtDeclareVariable(int,          viewpoint,                ,                   );
rtDeclareVariable(int,          cameraType,                ,                   );
rtDeclareVariable(uint,          useOrthoCamera,                ,                   );
rtDeclareVariable(float3,     shading_normal,    attribute shading_normal,    );



RT_PROGRAM void eyePassCamera() {
    //rtPrintf("in EPC\n");
    float3 rayOrigin;
    float3 rayDirection;
    float near, far;
    if(useOrthoCamera==1)
        rayOrtho(launchIndex, launchDim,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            cameraDir,
            nearPoint1, nearPoint2, nearPoint3, nearPoint4,
            rayOrigin, rayDirection, near, far);
    else
    {
      if(cameraType==1)
      {
        rayFisheye(
        //rayStandard(
              launchIndex, launchDim,
              eye, U, V, W,
              sceneCenter, sceneEpsilon, sceneMaxDim,
              rayOrigin, rayDirection, near, far);
       }
       else
       {
          rayStandard(
              launchIndex, launchDim,
              eye, U, V, W,
              sceneCenter, sceneEpsilon, sceneMaxDim,
              rayOrigin, rayDirection, near, far);
       }
    }
    optix::Ray ray(rayOrigin, rayDirection, eyeRayType, sceneEpsilon, far);

    EyePayload prd;
    prd.attenuation = make_float3(1);
    prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
    prd.depth       = 0;

    rtTrace(topObject, ray, prd);

}

// This is the camera for orthographic views (Such as rendering a wall
//   on the tabletop.  Necessary for both eye pass and importon pass
/*RT_PROGRAM void orthoEyePassCamera() {

    float3 rayOrigin;
    float3 rayDirection;
    float near, far;
    rayOrtho(launchIndex, launchDim,
            eye, U, V, W,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            rayOrigin, rayDirection, near, far);
    optix::Ray ray(rayOrigin, rayDirection, eyeRayType, near, far);

    EyePayload prd;
    prd.attenuation = make_float3(1);
    prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
    prd.depth       = 0;

    rtTrace(topObject, ray, prd);

}*/ 

RT_PROGRAM void eyePassCamera1D() {

  float3 centroid=centroidBuffer[launchIndex.x];
  float3 normal=normalBuffer[launchIndex.x];
  float3 rayDirection;
  float3 rayOrigin;

  rayOrigin =centroid+.1*normalize(normal);
  rayDirection = -1*normalize(normal);

  //Should we scale these better?
  const float dist =length(normal);
  const float near = max(sceneEpsilon, dist - sceneMaxDim / 2);
  const float far  =dist + sceneMaxDim;// / 2;

  optix::Ray ray(rayOrigin, rayDirection, eyeRayType, sceneEpsilon, far);

  EyePayload prd;
  prd.attenuation = make_float3(1);
  prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
  prd.depth       = 0;
  prd.centroidNumber=launchIndex.x;

  rtTrace(topObject, ray, prd);
}

//This has the logic for screens embedded in it.
RT_PROGRAM void shadowAnyHit() {
  const float3 direction = currentRay.direction;
  const float3 hitPoint  = currentRay.origin + hitT * direction;

  if(fmaxf(Ts) > 0) {
      float3 normal=normalBuffer[centroidNumber];
      if (screen)
      {
        int2 coords= screenCoord(hitPoint,normal, screenWidth,screenHeight);
        //rtPrintf("coords %d %d \n", coords.x, coords.y);
        if(screenBuffer[coords.y*screenWidth+coords.x]=='X')

            shadowPayload.attenuation=make_float3(0);
            //rtPrintf("blocked by screen\n");
      }
      //else
      //  rtPrintf("no  screen\n");
      shadowPayload.attenuation *= Ts;
      
      rtIgnoreIntersection();
      

  }
  // TODO: Fix logic
  else if(fmaxf(Kd) > 0) {
    shadowPayload.attenuation = make_float3(0);

    rtTerminateRay();
  }

}




//Importon hits (where we gather photons).  Direct light is going to be
//factored into a separate pass
RT_PROGRAM void eyeRayPassClosestHit() {
 //   rtPrintf("in eyePassClosestHit\n");
  float3 direction         = currentRay.direction;
  float3 hitPoint          = currentRay.origin + hitT * direction;
  float3 wsShadingNormal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
  float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
  float3 ffnormal          = faceforward(wsShadingNormal, -direction, wsGeometricNormal);

  float3 directIllumination = make_float3(0);

  if(fmaxf(Kd) > 0) {
    // Compute direct lighting
    int numLights = dirLightBuffer.size();
    while(numLights--)
    {
      const DirectionalLight& light = dirLightBuffer[numLights];


      float  nDL   = dot(ffnormal, -light.direction);


      if(nDL > 0)
      {
        ShadowPayload shadowPRD;
        shadowPRD.attenuation = make_float3(1);
        shadowPRD.lightIndex  = numLights;
        optix::Ray shadowRay(hitPoint, -light.direction, shadowRayType, sceneEpsilon);
        rtTrace(topShadower, shadowRay, shadowPRD);

        directIllumination =Kd * nDL * light.color* shadowPRD.attenuation*light.intensity;
        //if(directIllumination.x>0)
        //rtPrintf("direct Illum %f \n", directIllumination.x);
      } //end if nDl

    } //End while numlights--

    float distanceToEye = length(hitPoint - eye);

    //Just computing direct light... no hit
    float3 direct        = directIllumination;
    directBuffer[launchIndex]=direct;

    // eyePayload.attenuation = hit.Kd;
    eyePayload.direct = directIllumination;
  } // end if kd >0
  else
  {
    if(fmaxf(Ts) > 0)
    {

            

      eyePayload.attenuation=make_float3(0);
      eyePayload.attenuation *= Ts;
      
      //BEGIN HACK
      eyePayload.attenuation=make_float3(1);
      eyePayload.direct =make_float3(1);
      directBuffer[launchIndex]=make_float3(1);
      
      //++eyePayload.depth;

      //optix::Ray transmissionRay(hitPoint, direction, eyeRayType, sceneEpsilon, sceneMaxDim);
      //rtTrace(topObject, transmissionRay, eyePayload);


    } //end if fmaxf(Ts) > 0
  } //end else if kd >0
}


// We ignore backfacing polygons
RT_PROGRAM void eyeRayPassAnyHit() {
  if(backfaceCulling) {
    const float3 direction         = currentRay.direction;
    const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

    if(dot(direction, wsGeometricNormal) > 0) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void eyePassMiss()
{
  //rtPrintf("in eye pass miss\n");
  directBuffer[launchIndex]=make_float3(0,1,0);
  //eyePayload.direct=make_float3(30000);
}
