#include "hip/hip_runtime.h"
//importon pass
#include <optix.h>
#include <optix_math.h>

#include "Hit.hpp"
#include "Light.hpp"
#include "screen_utils.h"
#include "camera_rays.h"

struct EyePayload {
    int    centroidNumber;
  float3 attenuation;
  float3 direct;
  uint   depth;

};

struct ShadowPayload {
  float3 attenuation;
  uint   lightIndex;

};

rtBuffer<DirectionalLight> dirLightBuffer;
rtBuffer<HitRecord, 2>     eyeHitBuffer;
rtBuffer<float3,    1>     centroidBuffer;
rtBuffer<float3,    1>     normalBuffer;
rtBuffer<char,    1>     screenBuffer;

// Scene graph
rtDeclareVariable(rtObject,      topObject,       ,                           );
rtDeclareVariable(rtObject,      topShadower,     ,                           );

// Camera
rtDeclareVariable(float3,        eye,             ,                           );
rtDeclareVariable(float3,        U,               ,                           );
rtDeclareVariable(float3,        V,               ,                           );
rtDeclareVariable(float3,        W,               ,                           );

rtDeclareVariable(EyePayload,    eyePayload,      rtPayload,                  );
rtDeclareVariable(ShadowPayload, shadowPayload,   rtPayload,                  );
rtDeclareVariable(uint2,         launchIndex,     rtLaunchIndex,              );
rtDeclareVariable(uint2,         launchDim,       rtLaunchDim,                );
rtDeclareVariable(float3,        shadingNormal,   attribute shading_normal,   );
rtDeclareVariable(float3,        geometricNormal, attribute geometric_normal, );

rtDeclareVariable(uint,          importonType,    ,                           );
rtDeclareVariable(uint,          shadowRayType,   ,                           );
rtDeclareVariable(float3,        sceneCenter,     ,                           );
rtDeclareVariable(float,         sceneEpsilon,    ,                           );
rtDeclareVariable(float,         sceneMaxDim,     ,                           );
rtDeclareVariable(uint,          backfaceCulling, ,                           );

// Material properties
rtDeclareVariable(float3,        Kd,              ,                           );
rtDeclareVariable(float3,        Ks,              ,                           );
rtDeclareVariable(float3,        Td,              ,                           );
rtDeclareVariable(float3,        Ts,              ,                           );

rtDeclareVariable(float3,        nearPoint1,              ,                   );
rtDeclareVariable(float3,        nearPoint2,              ,                   );
rtDeclareVariable(float3,        nearPoint3,              ,                   );
rtDeclareVariable(float3,        nearPoint4,              ,                           );
rtDeclareVariable(float3,        cameraDir,               ,                   );
rtDeclareVariable(int,        screenWidth,                ,                   );
rtDeclareVariable(int,        screenHeight,               ,                   );
rtDeclareVariable(int,        screen,                     ,                   );

rtDeclareVariable(optix::Ray,    currentRay,      rtCurrentRay,               );
rtDeclareVariable(float,         hitT,            rtIntersectionDistance,     );
rtDeclareVariable(int,        centroidNumber,    attribute centroidNumber,    );
//debugging
rtDeclareVariable(int,          viewpoint,                ,                   );
rtDeclareVariable(float3,     shading_normal,    attribute shading_normal,    );
rtDeclareVariable(uint,          useOrthoCamera,                ,                   );
rtDeclareVariable(int,          cameraType,                ,                   );

//This file shouldn't need any knowledge of screens (no actual Lighting should be done)



//If we're shooting importons per triangle or patch
RT_PROGRAM void importonPassCamera1D() {

  float3 centroid=centroidBuffer[launchIndex.x];
  float3 normal=normalBuffer[launchIndex.x];

  float3 rayDirection;
  float3 rayOrigin;

  //If we're using the normal approach to finding the light on a patch
  if(viewpoint==0)
  {
      rayOrigin =centroid+.1*normalize(normal);
      rayDirection = -1*normalize(normal);
  }

  //if we're instead using rays shot from the eye to the centroid
  else
  {
      rayOrigin =eye;
      rayDirection = centroid-eye;
  }

  //Should we scale these better?
  const float dist =length(normal);
  const float near = max(sceneEpsilon, dist - sceneMaxDim / 2);
  const float far  =dist + sceneMaxDim;// / 2;

  optix::Ray ray(rayOrigin, rayDirection, importonType, sceneEpsilon, far);

  EyePayload prd;
  prd.attenuation    = make_float3(1);
  prd.direct         = make_float3(0.005f, 0.005f, 0.01f);
  prd.depth          = 0;
  prd.centroidNumber = launchIndex.x;

  rtTrace(topObject, ray, prd);
}

// This camera is for the placement of importons (or the points where we will
//   be gathering photons from.
RT_PROGRAM void importonPassCamera() {

    float3 rayOrigin;
    float3 rayDirection;
    float near;
    float far;

    if(useOrthoCamera==1)
        rayOrtho(launchIndex, launchDim,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            cameraDir,
            nearPoint1, nearPoint2, nearPoint3, nearPoint4,
            rayOrigin, rayDirection, near, far);
    else
    {
      if(cameraType==1)
        rayFisheye(launchIndex, launchDim,
            eye, U, V, W,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            rayOrigin, rayDirection, near, far);
      else
        rayStandard(launchIndex, launchDim,
            eye, U, V, W,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            rayOrigin, rayDirection, near, far);
    }
    optix::Ray ray(rayOrigin, rayDirection, importonType, sceneEpsilon, far);

    EyePayload prd;
    prd.attenuation = make_float3(1);
    prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
    prd.depth       = 0;

    rtTrace(topObject, ray, prd);

}


//When importons miss (we just display black for now).
RT_PROGRAM void importonPassMiss() {

  HitRecord hit;
  hit.flags = HitRecord::MISS;

  hit.Kd    = make_float3(0.000f, 0.000f, 0.00f);
  hit.flux  = make_float3(0);
  eyeHitBuffer[launchIndex] = hit;
  float3 normal=normalBuffer[launchIndex.x];

}

// We ignore backfacing polygons
RT_PROGRAM void importonPassAnyHit() {
  if(backfaceCulling) {
    const float3 direction         = currentRay.direction;
    const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

    if(dot(direction, wsGeometricNormal) > 0) {
      rtIgnoreIntersection();
    }
  }
}

//Importon hits (where we gather photons).  Direct light is going to be
//factored into a separate pass
RT_PROGRAM void importonPassClosestHit() {
  float3 direction         = currentRay.direction;
  float3 hitPoint          = currentRay.origin + hitT * direction;
  float3 wsShadingNormal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
  float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
  float3 ffnormal          = faceforward(wsShadingNormal, -direction, wsGeometricNormal);

  float3 directIllumination = make_float3(0);

  if(fmaxf(Kd) > 0) {

    //No light sampling needed

    float distanceToEye = length(hitPoint - eye);

    HitRecord hit;
    hit.position      = hitPoint;
    hit.normal        = ffnormal;
    hit.Kd            = Kd * eyePayload.attenuation;
    hit.flags         = HitRecord::HIT;
    hit.radiusSquared = powf(sceneMaxDim/4., 2);

    // TODO: Look for  reason for this number
    hit.minRdsSquared = powf(distanceToEye * tan(0.000136353848), 2);
    hit.photonCount   = 0;
    hit.flux          = make_float3(0);
    hit.direct        = directIllumination;

    eyeHitBuffer[launchIndex] = hit;
    //Direct light needs to be pulled out
    // eyePayload.attenuation = hit.Kd;
    //eyePayload.direct = directIllumination;
  } // end if kd >0
  else
  {
    if(fmaxf(Ts) > 0)
    {

      eyePayload.attenuation=make_float3(0);
      eyePayload.attenuation *= Ts;
      ++eyePayload.depth;

      optix::Ray transmissionRay(hitPoint, direction, importonType, sceneEpsilon, sceneMaxDim);
      rtTrace(topObject, transmissionRay, eyePayload);


    } //end if fmaxf(Ts) > 0
  } //end else if kd >0
}
