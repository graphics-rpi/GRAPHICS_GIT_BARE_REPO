#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

#include "Hit.hpp"
#include "Light.hpp"

struct EyePayload {
  float3 attenuation;
  float3 direct;
  uint   depth;
  int    centroidNumber;
};

struct ShadowPayload {
  float3 attenuation;
  uint   lightIndex;
};

// rtBuffer<PointLight>       lightBuffer;
rtBuffer<DirectionalLight> dirLightBuffer;
rtBuffer<HitRecord, 2>     eyeHitBuffer;
rtBuffer<float4,    2>     outputBuffer;

// Scene graph
rtDeclareVariable(rtObject,      topObject,       ,                           );
rtDeclareVariable(rtObject,      topShadower,     ,                           );

// Camera
rtDeclareVariable(float3,        eye,             ,                           );
rtDeclareVariable(float3,        U,               ,                           );
rtDeclareVariable(float3,        V,               ,                           );
rtDeclareVariable(float3,        W,               ,                           );

rtDeclareVariable(EyePayload,    eyePayload,      rtPayload,                  );
rtDeclareVariable(ShadowPayload, shadowPayload,   rtPayload,                  );
rtDeclareVariable(uint2,         launchIndex,     rtLaunchIndex,              );
rtDeclareVariable(uint2,         launchDim,       rtLaunchDim,                );
rtDeclareVariable(float3,        shadingNormal,   attribute shading_normal,   ); 
rtDeclareVariable(float3,        geometricNormal, attribute geometric_normal, );
rtDeclareVariable(float3,     centroid,          attribute centroid,   );
rtDeclareVariable(int,     centroidNumber,          attribute centroidNumber,   );

rtDeclareVariable(uint,          quickRayType,      ,                           );
rtDeclareVariable(uint,          quickShadowRayType,   ,                           );
rtDeclareVariable(float3,        sceneCenter,     ,                           );
rtDeclareVariable(float,         sceneEpsilon,    ,                           );
rtDeclareVariable(float,         sceneMaxDim,     ,                           );
rtDeclareVariable(uint,          backfaceCulling, ,                           );

// Material properties
rtDeclareVariable(float3,        Kd,              ,                           );
rtDeclareVariable(float3,        Ks,              ,                           );
rtDeclareVariable(float3,        Td,              ,                           );
rtDeclareVariable(float3,        Ts,              ,                           );

rtDeclareVariable(optix::Ray,    currentRay,      rtCurrentRay,               );
rtDeclareVariable(float,         hitT,            rtIntersectionDistance,     );

RT_PROGRAM void shadowAnyHit() {
  const float3 direction = currentRay.direction;
  const float3 hitPoint  = currentRay.origin + hitT * direction;

  // TODO: Fix logic
  if(fmaxf(Kd) > 0) {
    shadowPayload.attenuation = make_float3(0);

    rtTerminateRay();
  } else {
    if(fmaxf(Ts) > 0) {
      shadowPayload.attenuation *= Ts;
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void shadowMiss() {
  // const PointLight& light = dirLightBuffer[shadowPayload.lightIndex];
  // const float attenuatedDistance = max(hitT / light.attenuation, 1.0);
  // shadowPayload.attenuation = make_float3(1 / (attenuatedDistance * attenuatedDistance));
  // shadowPayload.attenuation = make_float3(0);
}

RT_PROGRAM void quickPass() {
  float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2 - 1;
  float3 rayOrigin = eye;
  float3 rayDirection = normalize(d.x * U + d.y * V + W);

  const float dist = length(eye - sceneCenter);
  const float near = max(sceneEpsilon, dist - sceneMaxDim / 2);
  const float far  = dist + sceneMaxDim / 2;

  optix::Ray ray(rayOrigin, rayDirection, quickRayType, sceneEpsilon, far);

  EyePayload prd;
  prd.attenuation = make_float3(1);
  prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
  prd.depth       = 0;

  rtTrace(topObject, ray, prd);
    rtPrintf("centroid number :-P %d", prd.centroidNumber);

   outputBuffer[launchIndex] = make_float4(prd.direct, 1.0);
}

RT_PROGRAM void quickPassMiss() {
  HitRecord hit;
  hit.flags = HitRecord::MISS;
  hit.Kd    = make_float3(0.005f, 0.005f, 0.01f);
  hit.flux  = make_float3(0);
  eyeHitBuffer[launchIndex] = hit;
}

RT_PROGRAM void eyePassAnyHit() {
  if(backfaceCulling) {
    const float3 direction         = currentRay.direction;
    const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

    if(dot(direction, wsGeometricNormal) < 0) {
      rtIgnoreIntersection();
    }
  }
}

RT_PROGRAM void eyePassClosestHit() {
  const float3 direction         = currentRay.direction;
  const float3 hitPoint          = currentRay.origin + hitT * direction;
  const float3 wsShadingNormal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
  const float3 wsGeometricNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));
  const float3 ffnormal          = faceforward(wsShadingNormal, -direction, wsGeometricNormal);

  float3 directIllumination = make_float3(0);

  if(fmaxf(Kd) > 0) {
    // Compute direct lighting
    int numLights = dirLightBuffer.size();
    while(numLights--) {
      const DirectionalLight& light = dirLightBuffer[numLights];

      // float3 L     = light.pos - hitPoint;
      // float  Ldist = length(L);
      // L /= Ldist;
      float  nDL   = dot(ffnormal, -light.direction);

      if(nDL > 0) {
        ShadowPayload shadowPRD;
        shadowPRD.attenuation = make_float3(1);
        shadowPRD.lightIndex  = numLights;
        optix::Ray shadowRay(hitPoint, -light.direction, quickShadowRayType, sceneEpsilon);
        rtTrace(topShadower, shadowRay, shadowPRD);

        directIllumination += Kd * nDL * light.color * shadowPRD.attenuation;
      }
    }

    float distanceToEye = length(hitPoint - eye);

    HitRecord hit;
    hit.position      = hitPoint;
    hit.normal        = ffnormal;
    hit.Kd            = Kd * eyePayload.attenuation;
    hit.flags         = HitRecord::HIT;
    // hit.radiusSquared = powf(distanceToEye * tan(0.000272707696 * 10), 2);
    // hit.radiusSquared = 1. / (100 * 100);
    // hit.radiusSquared = 1. / (4 * 4);
    // hit.radiusSquared = powf(2, 3);
    hit.radiusSquared = powf(2, 13);
    // TODO: Unmagify this number
    // hit.minRdsSquared = powf(distanceToEye * tan(0.000204530772), 2); // 45 degrees / 1920 / 2 in radians
    hit.minRdsSquared = powf(distanceToEye * tan(0.000136353848), 2); // 30 degrees / 1920 / 2 in radians
    // hit.minRdsSquared = powf(distanceToEye * tan(0.000227256413) * 0.25, 2); // 50 degrees / 1920 / 2 in radians
    // hit.minRdsSquared = powf(distanceToEye * tan(0.000272707696 * 1), 2); // 30 degrees / 1920 in radians
    hit.photonCount   = 0;
    hit.flux          = make_float3(0);
    hit.direct        = directIllumination;

    eyeHitBuffer[launchIndex] = hit;
    // eyePayload.attenuation = hit.Kd;
    eyePayload.direct = directIllumination;
  } else {
    if(fmaxf(Ts) > 0) {
      eyePayload.attenuation *= Ts;
      ++eyePayload.depth;

      optix::Ray transmissionRay(hitPoint, direction, quickRayType, sceneEpsilon, sceneMaxDim);
      rtTrace(topObject, transmissionRay, eyePayload);
//        eyePayload.centroidNumber=centroidNumber;


    }
  }
}
