#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "Hit.hpp"
#include "Light.hpp"

using namespace optix;
rtDeclareVariable(float3,     centroid,          attribute centroid,   );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3,        Kd,              ,                           );
rtDeclareVariable(optix::Ray,    currentRay,      rtCurrentRay,               );
rtDeclareVariable(int,     centroidNumber,          attribute centroidNumber,   );
rtDeclareVariable(int,     patch,          attribute patch,   );
rtDeclareVariable(float,     gammaAttr,          attribute gammaAttr,   );
rtDeclareVariable(float,     betaAttr,          attribute betaAttr,   );
rtDeclareVariable(rtObject,      topObject,       ,                           );
rtDeclareVariable(float3,        Ts,              ,                           );
rtDeclareVariable(float,         hitT,            rtIntersectionDistance,     );
rtDeclareVariable(uint,          quickRayType,      ,                           );
rtDeclareVariable(float,         sceneEpsilon,    ,                           );
rtDeclareVariable(float,         sceneMaxDim,     ,                           );
rtDeclareVariable(float3,     rayDirection,          attribute rayDirection,   );


struct PerRayData_radiance
{
  float3 attenuation;
  float3 hitPoint;
  int    centroidNumber;

  float3 direct;
  uint   depth;
  int patch;
  float beta;
  float gamma;
  

};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );


RT_PROGRAM void any_hit_shadow()
{
  //if(fmaxf(Ts) > 0)
  //  rtIgnoreIntersection();

  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(.0);

  rtTerminateRay();
}



RT_PROGRAM void any_hit()
{
  if(dot(geometric_normal,rayDirection)>0)
    rtIgnoreIntersection();
}
RT_PROGRAM void closest_hit()
{

  const float3 direction         = currentRay.direction;
  const float3 hitPoint          = currentRay.origin + hitT * direction;

    if(fmaxf(Ts) > 0) {
    prd_radiance.centroidNumber=-1;
    /*  //currentRay.attenuation *= Ts;
      //++currentRay.depth;
      
      optix::Ray transmissionRay(hitPoint, direction, quickRayType, sceneEpsilon, sceneMaxDim);
      rtTrace(topObject, transmissionRay, prd_radiance);*/
  }
  else
  {
    prd_radiance.centroidNumber=centroidNumber;
  }
    prd_radiance.attenuation =normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    prd_radiance.patch=patch;
    prd_radiance.beta=betaAttr;
    prd_radiance.gamma=gammaAttr;
    prd_radiance.hitPoint=hitPoint;
//  }
  //prd_radiance.attenuation =make_float3(dot(normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)),currentRay.direction));
  
  //normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));//make_float3(0,0,1.);//centroid;//Kd*make_float3(dot(normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)),currentRay.direction)*0.5f + 0.5f);
//prd_radiance.result = make_float3(1,1,1);
  //rtPrintf("Centroid nubmer %d\n", prd_radiance.centroidNumber);
}

RT_PROGRAM void miss() {
  prd_radiance.attenuation= make_float3(0.005f, 0.005f, 1.f);
  prd_radiance.centroidNumber=-1;

}
