#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

#include "Hit.hpp"
#include "Photon.hpp"
#include "Light.hpp"
#include "num_neighbors.h"
#include "camera_rays.h"
#define ALPHA 0.5
//#define OMIT_FIRST

struct EyePayload {
  float3 attenuation;
  float3 hitPoint;
  int centroidNumber;
  float3 direct;
  uint   depth;
  int patch;
  float beta;
  float gamma;

};
rtBuffer<float> MomentBuffer;
rtBuffer<PackedHitRecord, 2> eyeHitBuffer;
rtBuffer<float4,          2> outputBuffer;
rtBuffer<float4,          1> patchValueBuffer;
rtBuffer<float3,          1> centroidBuffer;
rtBuffer<float,           1> patchAreaBuffer;
rtBuffer<int,             1> patchSizesBuffer;
rtBuffer<int,             1> patchStartIndexesBuffer;
rtBuffer<float,           1> triAreasBuffer;
rtBuffer<int,             1> neighborsBuffer;
rtBuffer<float,           1> neighborWeightsBuffer;
rtBuffer<float3,          2> directBuffer;
rtBuffer<float4,          1> intermediateBuffer;

rtDeclareVariable(uint,          numTriangles,      ,                         );
rtDeclareVariable(uint2,         launchDim,       rtLaunchDim,                );
rtDeclareVariable(rtObject,      topObject,       ,                           );

rtDeclareVariable(uint2,         launchIndex,       rtLaunchIndex,            );
rtDeclareVariable(int,           centroidNumber,          attribute centroidNumber, );
rtDeclareVariable(int,           numPatches,          ,                             ) ;
rtDeclareVariable(float3,        sceneCenter,     ,                           );
rtDeclareVariable(float,         sceneEpsilon,    ,                           );
rtDeclareVariable(float,         sceneMaxDim,     ,                           );
rtDeclareVariable(uint,          quickRayType,      ,                           );
rtDeclareVariable(int,           res,      ,                           );
rtDeclareVariable(int,           NUM_MOMENTS,      ,                           );
rtDeclareVariable(EyePayload,    eyePayload,      rtPayload,                  );


rtDeclareVariable(float3,     shading_normal,    attribute shading_normal,   );

// Camera
rtDeclareVariable(float3,        eye,             ,                           );
rtDeclareVariable(float3,        U,               ,                           );
rtDeclareVariable(float3,        V,               ,                           );
rtDeclareVariable(float3,        W,               ,                           );
rtDeclareVariable(float3,        cameraDir,               ,                   );
rtDeclareVariable(float3,        nearPoint1,              ,                   );
rtDeclareVariable(float3,        nearPoint2,              ,                   );
rtDeclareVariable(float3,        nearPoint3,              ,                   );
rtDeclareVariable(float3,        nearPoint4,              ,                           );
rtDeclareVariable(uint,          useOrthoCamera,                ,                   );
rtDeclareVariable(float,         toobright,                ,                   );
rtDeclareVariable(float,         toodim,                ,                   );
rtDeclareVariable(int,         greyscale,                ,                   );
rtDeclareVariable(int,          cameraType,                ,                   );




RT_PROGRAM void preRegatherPass() {
  // rtPrintf("regather pass\n");

    float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2 - 1;
    float3 rayOrigin = eye;
    float3 rayDirection = normalize(d.x * U + d.y * V + W);

    const float dist = length(eye - sceneCenter);
    const float near = max(sceneEpsilon, dist - sceneMaxDim / 2);
    const float far  = dist + sceneMaxDim / 2;

    optix::Ray ray(rayOrigin, rayDirection, quickRayType, sceneEpsilon, far);

      EyePayload prd;
  prd.attenuation = make_float3(1);
  prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
  prd.depth       = 0;

  rtTrace(topObject, ray, prd);
  int cent=prd.centroidNumber;



}

RT_PROGRAM void patchGatherPass()
{
   int patchNum=launchIndex.x;
   float4 tempor=make_float4(0);
    //rtPrintf("num_tris %d \n", num_tris);
   int num_tris=patchSizesBuffer[patchNum];
   int triNum=patchStartIndexesBuffer[patchNum];
   float triArea;
   float totalArea=0.f;
   for (int i=0; i<num_tris; i++,triNum++)
    {
      triArea=triAreasBuffer[triNum];
      tempor+=intermediateBuffer[triNum]*triArea/patchAreaBuffer[patchNum];
    }
    patchValueBuffer[patchNum]=tempor;
}

RT_PROGRAM void regatherPass() {

    float4 outputVal=make_float4(directBuffer[launchIndex],1);

    float3 rayOrigin;
    float3 rayDirection;
    float near, far;
    if(useOrthoCamera==1)
        rayOrtho(launchIndex, launchDim,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            cameraDir,
            nearPoint1, nearPoint2, nearPoint3, nearPoint4,
            rayOrigin, rayDirection, near, far);
    else
    {
      if(cameraType==1)
      rayFisheye(
            launchIndex, launchDim,
            eye, U, V, W,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            rayOrigin, rayDirection, near, far);
      else
      rayStandard(
            launchIndex, launchDim,
            eye, U, V, W,
            sceneCenter, sceneEpsilon, sceneMaxDim,
            rayOrigin, rayDirection, near, far);
    }
    optix::Ray ray(rayOrigin, rayDirection, quickRayType, sceneEpsilon, far);

    EyePayload prd;
    prd.attenuation = make_float3(1);
    prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
    prd.depth       = 0;

    rtTrace(topObject, ray, prd);
    int cent=prd.centroidNumber;



  if(cent==-1) //This is very bad
  {
          bool dark=false;
        bool bright=false;
        if(outputVal.x>toobright)
          bright=true;
        if(outputVal.x<toodim)
          dark=true;
        if(bright||dark)
        {
          PackedHitRecord hit = eyeHitBuffer[launchIndex];
          float3 hitPosition  = prd.hitPoint+make_float3(500);//make_float3(hit.a);
          //float hit_sum=(hitPosition.x+hitPosition.y+hitPosition.z)*50;
          //int hit_sum_int=(int)hit_sum;
          int count=0;
          if( abs(hitPosition.x*50.f-(int)(hitPosition.x*50))>.5 )
              count++;
          if( abs(hitPosition.y*50.f-(int)(hitPosition.y*50))>.5 )
              count++;
          if( abs(hitPosition.z*50.f-(int)(hitPosition.z*50))>.5 )
              count++;

          if(count%2==0)
          {
            if(bright)
              outputVal=make_float4(outputVal.x*1.6,
                                    outputVal.y*1.4,
                                    outputVal.z*0.4,
                                    outputVal.w);
            else //dark
                          outputVal=make_float4(outputVal.x*0.6666,
                                    outputVal.y*0.6666,
                                    outputVal.z*1.5,
                                    outputVal.w);

          }//end ifcount
          //else
          //  outputBuffer[launchIndex]=make_float4(0,1,0,1);
        }//end if bright||dark
      //outputBuffer[launchIndex] = outputVal; //make_float4(0.0,1.0,0,1);//   intermediateBuffer[eyePayload.centroidNumber];
      outputBuffer[launchIndex] = make_float4(0.0,0.0,0.0,1);//   intermediateBuffer[eyePayload.centroidNumber];
  }
  else
  {

    if (res==0)
    {
      //outputBuffer[launchIndex] =  intermediateBuffer[launchIndex];
    }
    else if(res==1)
    {
        float4 tempor=make_float4(0);
        int num_tris=patchSizesBuffer[prd.patch];
        int triNum=patchStartIndexesBuffer[prd.patch];
        outputBuffer[launchIndex] =  patchValueBuffer[prd.patch];
    }

    else
    {

       if(res==3)//hybrid
       {
            PackedHitRecord hit = eyeHitBuffer[launchIndex];
            const float3 hitPosition  = make_float3(hit.a);
            float        hitRadiusSq  = hit.a.w;

            const float3 hitNormal    = make_float3(hit.b);
            float        hitPhotonCnt = hit.b.w;
            const float3 hitKd        = make_float3(hit.c);
            const float  hitMinRds    = hit.c.w;
            float3       hitFlux      = make_float3(hit.d);
            const float3 hitDirect    = make_float3(hit.e);
            const uint   hitFlags     = __float_as_int(hit.e.w);
            //if(!(hitFlags & HitRecord::HIT))
            {
              outputBuffer[launchIndex] = make_float4(.5*directBuffer[launchIndex], 1);
            }
            //else
            //  outputBuffer[launchIndex]=make_float4(0,1,0,1);
       }//end if hybrid
       else//Just tris
       {
          outputBuffer[launchIndex]=make_float4(0,0,0,1);
       }

        float beta=prd.beta;
        float gamma=prd.gamma;
        //outputBuffer[launchIndex] =  intermediateBuffer[prd.centroidNumber];// make_float
        int index = prd.centroidNumber;
        float4 val0=make_float4(0);
        float4 val1=make_float4(0);
        float4 val2=make_float4(0);
        float neighborWeightSum0=0;
        float neighborWeightSum1=0;
        float neighborWeightSum2=0;
        int num_neighbors=NUM_NEIGHBORS;
        //If we are doing non-interpolated tris;
        if(res==5)
        {
          num_neighbors=1;
          beta=gamma=1.f/3.f;
        }

        //Triangle interpolation
        for(int i =0; i< num_neighbors; i++)
        {
            //int index=launchIndex.x*8*3;
            int index0=(3*index+0)*NUM_NEIGHBORS+i;
            int index1=(3*index+1)*NUM_NEIGHBORS+i;
            int index2=(3*index+2)*NUM_NEIGHBORS+i;
            float neighborWeight0=neighborWeightsBuffer[index0];
            float neighborWeight1=neighborWeightsBuffer[index1];
            float neighborWeight2=neighborWeightsBuffer[index2];
            #ifdef OMIT_FIRST
            if(i!=0)
            #endif
            {
                neighborWeightSum0+=neighborWeight0;
                neighborWeightSum1+=neighborWeight1;
                neighborWeightSum2+=neighborWeight2;
                val0 += intermediateBuffer[neighborsBuffer[index0]] * neighborWeight0;
                val1 += intermediateBuffer[neighborsBuffer[index1]] * neighborWeight1;
                val2 += intermediateBuffer[neighborsBuffer[index2]] * neighborWeight2;
            }

        }//endfor
        float4 outputVal=outputBuffer[launchIndex]
                                     + (1.0f - beta - gamma) *  val0/ neighborWeightSum0
                                     +  gamma              *  val2 / neighborWeightSum2
                                     +  beta               * val1 / neighborWeightSum1 ;
        if(greyscale==1)
        {
          //rtPrintf("greyscaling\n");
          float temp=(outputVal.x+outputVal.y+outputVal.z)/3.f;
          float tempa=outputVal.w;
          if(temp>1) temp=1;
          temp=.2+.6*temp;
          outputVal=make_float4(temp,temp,temp, tempa);
        }//end ifgreyscale
        bool dark=false;
        bool bright=false;
        if(outputVal.x>toobright)
          bright=true;
        if(outputVal.x<toodim)
          dark=true;
        if(bright||dark)
        {
          PackedHitRecord hit = eyeHitBuffer[launchIndex];
          float3 hitPosition  = prd.hitPoint+make_float3(500);//make_float3(hit.a);
          //float hit_sum=(hitPosition.x+hitPosition.y+hitPosition.z)*50;
          //int hit_sum_int=(int)hit_sum;
          int count=0;
          if( abs(hitPosition.x*50.f-(int)(hitPosition.x*50))>.5 )
              count++;
          if( abs(hitPosition.y*50.f-(int)(hitPosition.y*50))>.5 )
              count++;
          if( abs(hitPosition.z*50.f-(int)(hitPosition.z*50))>.5 )
              count++;

          if(count%2==0)
          {
            if(bright)
              outputVal=make_float4(outputVal.x*1.5,
                                    outputVal.y*0.6666,
                                    outputVal.z*0.6666,
                                    outputVal.w);
            else //dark
                          outputVal=make_float4(outputVal.x*0.6666,
                                    outputVal.y*0.6666,
                                    outputVal.z*1.5,
                                    outputVal.w);

          }//end ifcount
          //else
        }//end if bright||dark
        outputBuffer[launchIndex]=outputVal;
      }//end not res 0 or res 1
    }//end big else


}

RT_PROGRAM void momentGatherPass() {

    float2 d = make_float2(launchIndex) / make_float2(launchDim) * 2 - 1;
    float3 rayOrigin = eye;
    float3 rayDirection = normalize(d.x * U + d.y * V + W);

    const float dist = length(eye - sceneCenter);
    const float near = max(sceneEpsilon, dist - sceneMaxDim / 2);
    const float far  = dist + sceneMaxDim / 2;

    optix::Ray ray(rayOrigin, rayDirection, quickRayType, sceneEpsilon, far);

    EyePayload prd;
    prd.attenuation = make_float3(1);
    prd.direct      = make_float3(0.005f, 0.005f, 0.01f);
    prd.depth       = 0;

    rtTrace(topObject, ray, prd);
    int cent=prd.centroidNumber;


  if(cent==-1) //This is very bad
  {
      outputBuffer[launchIndex] =   make_float4(0.0,0.0,0.0,1);//   intermediateBuffer[eyePayload.centroidNumber];
  }
  else
  {
        int index = prd.centroidNumber;
        float max=0;
        float temp;

        for(int i=0; i<NUM_MOMENTS; i++)
        {
          temp=MomentBuffer[i*numTriangles+index];
          if (temp>max)
            max=temp;
        }

        outputBuffer[launchIndex] =  make_float4(max);


    }


}
