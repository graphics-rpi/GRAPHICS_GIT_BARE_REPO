#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_math.h>

#include "Hit.hpp"
#include "Photon.hpp"

rtBuffer<uint>               windowHitBuffer;
rtBuffer<PackedPhotonRecord> photonMapBuffer;
rtBuffer<PackedHitRecord, 2> eyeHitBuffer;
rtBuffer<float4,          2> outputBuffer;

rtDeclareVariable(uint,          totalPhotonsFired, ,                           );
rtDeclareVariable(uint2,         launchIndex,       rtLaunchIndex,              );

__device__ __inline__ 
void accumulatePhoton(
  const PackedPhotonRecord& photon,
  const float3& hitNormal,
  const float3& hitKd,
  uint& numNewPhotons, float3& totalFlux
) {
  float3 photonEnergy = make_float3(photon.d);
  float3 photonNormal = make_float3(photon.b);
  float  cos = dot(photonNormal, hitNormal);
  if(cos > 0.01f) { // Fudge factor for imperfect cornell box geom
    windowHitBuffer[__float_as_int(photon.a.w)] = 1;
    float3 photonDir = make_float3(photon.c);
    float3 flux = photonEnergy * hitKd * -dot(photonDir, hitNormal);
    ++numNewPhotons;
    totalFlux += flux;
  }
}

RT_PROGRAM void gatherPass() {
  PackedHitRecord hit = eyeHitBuffer[launchIndex];
  const float3 hitPosition  = make_float3(hit.a);
  const float  hitRadiusSq  = hit.a.w;
  const float3 hitNormal    = make_float3(hit.b);
  const float  hitPhotonCnt = hit.b.w;
  const float3 hitKd        = make_float3(hit.c);
  const float3 hitFlux      = make_float3(hit.d);
  const float3 hitDirect    = make_float3(hit.e);
  const uint   hitFlags     = __float_as_int(hit.e.w);

  // TODO: Confirm this does what we think we want it to do
  if(!(hitFlags & HitRecord::HIT)) {
    outputBuffer[launchIndex] = make_float4(hitKd, 1);
    return;
  }

#define MAX_DEPTH 20
  unsigned int stack[MAX_DEPTH];
  unsigned int stackSize = 0;
  unsigned int node      = 0;

#define pushNode(N) stack[stackSize++] = (N)
#define popNode()   stack[--stackSize]

  float3 flux = make_float3(0);

  pushNode(0);
  unsigned int numNewPhotons = 0;
  unsigned int iterations    = 0;

  do {
    const PackedPhotonRecord& photon = photonMapBuffer[node];

    unsigned int axis = __float_as_int(photon.d.w);
    if(!(axis & PhotonRecord::NILL)) {
      float3 photonPosition = make_float3(photon.a);
      float3 diff = hitPosition - photonPosition;
      float  distanceSq = dot(diff, diff);

      if(distanceSq <= hitRadiusSq) {
        accumulatePhoton(photon, hitNormal, hitKd, numNewPhotons, flux);
      }

      if(!(axis & PhotonRecord::LEAF)) {
        float d;
        if     (axis & PhotonRecord::X) d = diff.x;
        else if(axis & PhotonRecord::Y) d = diff.y;
        else                            d = diff.z;

        int child = d < 0 ? 0 : 1;
        if(d * d < hitRadiusSq) {
          pushNode((node << 1) + 2 - child);
        }
        node = (node << 1) + 1 + child;
      } else {
        node = popNode();
      }
    } else {
      node = popNode();
    }
    ++iterations;
  } while(node);

#define ALPHA 0.6

  const float n = hitPhotonCnt;
  const float m = static_cast<float>(numNewPhotons);

  const float nHat      = n + ALPHA * m;
  const float reduction = m > 0 ? nHat / (n + m) : 1;
  const float rHatSq    = hitRadiusSq * reduction;

  const float3 tauNHat  = (hitFlux + flux) * reduction;

  hit.a.w = rHatSq;
  hit.b.w = nHat;

  hit.d   = make_float4(tauNHat);
  eyeHitBuffer[launchIndex] = hit;

  float3 indirectFlux = 1.0 / (M_PI * hitRadiusSq) * (tauNHat / totalPhotonsFired);
  outputBuffer[launchIndex] = make_float4(hitDirect + indirectFlux * 1000000);
}
