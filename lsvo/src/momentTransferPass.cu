#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>



rtDeclareVariable(uint,        curMoment,             ,                           );
rtBuffer<float4,          2> outputBuffer;
rtBuffer<float4,          1> intermediateBuffer;
rtBuffer<float> MomentBuffer;

rtDeclareVariable(uint,          numFramesPerMoment,      ,                           );
rtDeclareVariable(uint,          numTriangles,      ,                           );
rtDeclareVariable(uint2,         launchIndex,       rtLaunchIndex,              );

RT_PROGRAM void momentTransferPass()
{
  //uint2 index=make_uint2(launchIndex.x+curMoment*numFramesPerMoment,0);
  //float4 temp = intermediateBuffer[index];
  //float4 temp = intermediateBuffer[launchIndex.x+curMoment*numFramesPerMoment];
  float4 temp = intermediateBuffer[launchIndex.x];
  float tempfloat=0.2989*temp.x + 0.5870*temp.y+0.1140*temp.z;
  //rtPrintf("numTriangles %d curMomeent %d \n", numTriangles, curMoment);
  MomentBuffer[curMoment*numTriangles+launchIndex.x]= tempfloat;
}
