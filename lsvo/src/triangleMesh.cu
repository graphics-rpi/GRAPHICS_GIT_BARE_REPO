#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix.h>
#include <optixu/optixu_aabb.h>

// This is to be plugged into an RTgeometry object to represent
// a triangle mesh with a vertex buffer of triangle soup (triangle list)
// with an interleaved position, normal, texturecoordinate layout.

rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<float2> texcoordBuffer;
rtBuffer<int3>   vertexIndexBuffer;    // position indices
rtBuffer<int3>   normalIndexBuffer;    // normal indices
rtBuffer<int3>   texcoordIndexBuffer;  // texcoord indices
rtBuffer<int>   centroidIndexBuffer;  // texcoord indices
rtBuffer<int>   patchIndexBuffer;  // texcoord indices
rtBuffer<uint>   materialBuffer;       // per-face material index

rtDeclareVariable(float3,     texcoord,          attribute texcoord,         );
rtDeclareVariable(float3,     geometric_normal,  attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,    attribute shading_normal,   );

rtDeclareVariable(float3,     centroid,          attribute centroid,   );
rtDeclareVariable(float3,     rayDirection,          attribute rayDirection,   );
rtDeclareVariable(int,        patch,             attribute patch,   );
rtDeclareVariable(int,        centroidNumber,    attribute centroidNumber,   );
rtDeclareVariable(int,        triNumber,    attribute triNumber,   );
//rtDeclareVariable(int,        isGatherPass , ,   );
rtDeclareVariable(float,        betaAttr , attribute betaAttr,   );
rtDeclareVariable(float,        gammaAttr , attribute gammaAttr,   );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay,                             );

RT_PROGRAM void meshIntersect(int primIdx) {
  //rtPrintf("intersected with %i \n", primIdx);
  int3 v_idx = vertexIndexBuffer[primIdx];

  float3 p0 = vertexBuffer[v_idx.x];
  float3 p1 = vertexBuffer[v_idx.y];
  float3 p2 = vertexBuffer[v_idx.z];
	// centroid= (p0+p1+p2)/3.;
	// centroidNumber=centroidIndexBuffer[primIdx];
	// patch=patchIndexBuffer[primIdx];
	// triNumber=primIdx;
  //centroid=centroid/3.;
  // Intersect ray with triangle
  float3 e0 = p1 - p0;
  float3 e1 = p0 - p2;
  float3 n  = cross(e0, e1);

  float v   = dot(n, ray.direction);

  float r   = 1.0f / v;

  float3 e2 = p0 - ray.origin;
  float va  = dot(n, e2);
  float t   = r * va;

  if(t < ray.tmax && t > ray.tmin) {
    float3 i   = cross(e2, ray.direction);
    float v1   = dot(i, e1);
    float beta = r * v1;
    if(beta >= 0.0f){
      float v2 = dot(i, e0);
      float gamma = r * v2;
      if((v1 + v2) * v <= v * v && gamma >= 0.0f) {
        if(rtPotentialIntersection(t)) {
					rayDirection=ray.direction;
					centroid= (p0+p1+p2)/3.;
					centroidNumber=centroidIndexBuffer[primIdx];
					patch=patchIndexBuffer[primIdx];
					triNumber=primIdx;

          int3 n_idx = normalIndexBuffer[primIdx];

          if(normalBuffer.size() == 0 || n_idx.x < 0 || n_idx.y < 0 || n_idx.z < 0) {
            shading_normal = -n;
          } else {
            float3 n0 = normalBuffer[n_idx.x];
            float3 n1 = normalBuffer[n_idx.y];
            float3 n2 = normalBuffer[n_idx.z];
            shading_normal = normalize(n1 * beta + n2 * gamma + n0 * (1.0f - beta - gamma));
          }
          geometric_normal = -n;

          int3 t_idx = texcoordIndexBuffer[primIdx];
          if(texcoordBuffer.size() == 0 || t_idx.x < 0 || t_idx.y < 0 || t_idx.z < 0) {
            texcoord = make_float3(0.0f, 0.0f, 0.0f);
          } else {
            float2 t0 = texcoordBuffer[t_idx.x];
            float2 t1 = texcoordBuffer[t_idx.y];
            float2 t2 = texcoordBuffer[t_idx.z];
            texcoord = make_float3(t1 * beta + t2 * gamma + t0 * (1.0f - beta - gamma));
          }
          //rtPrintf("Centroid nubmer %d\n", centroidNumber);
          //if(!isGatherPass||v>0)

          betaAttr=beta;
          gammaAttr=gamma;
          rtReportIntersection(materialBuffer[primIdx]);
        }
      }
    }
  }
}

RT_PROGRAM void meshBounds(int primIdx, float result[6]) {
  int3 v_idx = vertexIndexBuffer[primIdx];

  float3 v0 = vertexBuffer[v_idx.x];
  float3 v1 = vertexBuffer[v_idx.y];
  float3 v2 = vertexBuffer[v_idx.z];

  optix::Aabb* aabb = (optix::Aabb*)result;
  aabb->m_min = fminf(fminf(v0, v1), v2);
  aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
}
